// Copyright 2019-2025, Relay Therapeutics
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "assert.h"
#include "centroid_restraint.hpp"
#include "gpu_utils.cuh"
#include "k_centroid_restraint.cuh"
#include "math_utils.cuh"
#include <vector>

namespace tmd {

template <typename RealType>
CentroidRestraint<RealType>::CentroidRestraint(
    const std::vector<int> &group_a_idxs, const std::vector<int> &group_b_idxs,
    const RealType kb, const RealType b0)
    : N_A_(group_a_idxs.size()), N_B_(group_b_idxs.size()), kb_(kb), b0_(b0) {

  cudaSafeMalloc(&d_group_a_idxs_, N_A_ * sizeof(*d_group_a_idxs_));
  gpuErrchk(hipMemcpy(d_group_a_idxs_, &group_a_idxs[0],
                       N_A_ * sizeof(*d_group_a_idxs_),
                       hipMemcpyHostToDevice));

  cudaSafeMalloc(&d_group_b_idxs_, N_B_ * sizeof(*d_group_b_idxs_));
  gpuErrchk(hipMemcpy(d_group_b_idxs_, &group_b_idxs[0],
                       N_B_ * sizeof(*d_group_b_idxs_),
                       hipMemcpyHostToDevice));

  cudaSafeMalloc(&d_centroid_a_, 3 * sizeof(*d_centroid_a_));
  cudaSafeMalloc(&d_centroid_b_, 3 * sizeof(*d_centroid_b_));
};

template <typename RealType> CentroidRestraint<RealType>::~CentroidRestraint() {
  gpuErrchk(hipFree(d_group_a_idxs_));
  gpuErrchk(hipFree(d_group_b_idxs_));
  gpuErrchk(hipFree(d_centroid_a_));
  gpuErrchk(hipFree(d_centroid_b_));
};

template <typename RealType>
void CentroidRestraint<RealType>::execute_device(
    const int batches, const int N, const int P, const RealType *d_x,
    const RealType *d_p, const RealType *d_box, unsigned long long *d_du_dx,
    unsigned long long *d_du_dp,
    __int128 *d_u, // [1]
    hipStream_t stream) {

  assert(batches == 1);
  if (N_B_ + N_A_ > 0) {
    int tpb = DEFAULT_THREADS_PER_BLOCK;

    int blocks = ceil_divide(N_B_ + N_A_, tpb);
    gpuErrchk(hipMemsetAsync(d_centroid_a_, 0.0, 3 * sizeof(*d_centroid_a_),
                              stream));
    gpuErrchk(hipMemsetAsync(d_centroid_b_, 0.0, 3 * sizeof(*d_centroid_b_),
                              stream));
    k_calc_centroid<RealType>
        <<<blocks, tpb, 0, stream>>>(d_x, d_group_a_idxs_, d_group_b_idxs_,
                                     N_A_, N_B_, d_centroid_a_, d_centroid_b_);
    gpuErrchk(hipPeekAtLastError());

    k_centroid_restraint<RealType><<<blocks, tpb, 0, stream>>>(
        d_x, d_group_a_idxs_, d_group_b_idxs_, N_A_, N_B_, d_centroid_a_,
        d_centroid_b_, kb_, b0_, d_du_dx,
        d_u // Can write directly to the energy buffer for this potential.
    );
    gpuErrchk(hipPeekAtLastError());
  }
};

template class CentroidRestraint<double>;
template class CentroidRestraint<float>;

} // namespace tmd
