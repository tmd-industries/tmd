// Copyright 2019-2025, Relay Therapeutics
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "chiral_bond_restraint.hpp"
#include "gpu_utils.cuh"
#include "k_chiral_restraint.cuh"
#include "kernel_utils.cuh"
#include "math_utils.cuh"
#include <hipcub/hipcub.hpp>
#include <vector>

namespace tmd {

template <typename RealType>
ChiralBondRestraint<RealType>::ChiralBondRestraint(
    const std::vector<int> &idxs, const std::vector<int> &signs)
    : R_(idxs.size() / 4), sum_storage_bytes_(0),
      kernel_ptrs_({// enumerate over every possible kernel combination
                    // U: Compute U
                    // X: Compute DU_DX
                    // P: Compute DU_DP                U  X  P
                    &k_chiral_bond_restraint<RealType, 0, 0, 0>,
                    &k_chiral_bond_restraint<RealType, 0, 0, 1>,
                    &k_chiral_bond_restraint<RealType, 0, 1, 0>,
                    &k_chiral_bond_restraint<RealType, 0, 1, 1>,
                    &k_chiral_bond_restraint<RealType, 1, 0, 0>,
                    &k_chiral_bond_restraint<RealType, 1, 0, 1>,
                    &k_chiral_bond_restraint<RealType, 1, 1, 0>,
                    &k_chiral_bond_restraint<RealType, 1, 1, 1>}) {

  if (idxs.size() % 4 != 0) {
    throw std::runtime_error("idxs.size() must be exactly 4*R!");
  }

  if (R_ != signs.size()) {
    throw std::runtime_error("signs.size() must be exactly R!");
  }

  for (auto s : signs) {
    if (s != -1 && s != 1) {
      throw std::runtime_error(
          "signs must be comprised exclusively of 1 or -1");
    }
  }

  cudaSafeMalloc(&d_idxs_, R_ * 4 * sizeof(*d_idxs_));
  gpuErrchk(hipMemcpy(d_idxs_, &idxs[0], R_ * 4 * sizeof(*d_idxs_),
                       hipMemcpyHostToDevice));

  cudaSafeMalloc(&d_signs_, R_ * sizeof(*d_signs_));
  gpuErrchk(hipMemcpy(d_signs_, &signs[0], R_ * sizeof(*d_signs_),
                       hipMemcpyHostToDevice));

  cudaSafeMalloc(&d_u_buffer_, R_ * sizeof(*d_u_buffer_));

  gpuErrchk(hipcub::DeviceReduce::Sum(nullptr, sum_storage_bytes_, d_u_buffer_,
                                   d_u_buffer_, R_));

  gpuErrchk(hipMalloc(&d_sum_temp_storage_, sum_storage_bytes_));
};

template <typename RealType>
ChiralBondRestraint<RealType>::~ChiralBondRestraint() {
  gpuErrchk(hipFree(d_idxs_));
  gpuErrchk(hipFree(d_signs_));
  gpuErrchk(hipFree(d_u_buffer_));
  gpuErrchk(hipFree(d_sum_temp_storage_));
};

template <typename RealType>
void ChiralBondRestraint<RealType>::execute_device(
    const int batches, const int N, const int P, const RealType *d_x,
    const RealType *d_p, const RealType *d_box, unsigned long long *d_du_dx,
    unsigned long long *d_du_dp, __int128 *d_u, hipStream_t stream) {

  assert(batches == 1);
  if (P != R_) {
    throw std::runtime_error(
        "ChiralBondRestraint::execute_device(): expected P == R, got P=" +
        std::to_string(P) + ", R=" + std::to_string(R_));
  }

  if (R_ > 0) {
    const int tpb = DEFAULT_THREADS_PER_BLOCK;
    const int blocks = ceil_divide(R_, tpb);

    int kernel_idx = 0;
    kernel_idx |= d_du_dp ? 1 << 0 : 0;
    kernel_idx |= d_du_dx ? 1 << 1 : 0;
    kernel_idx |= d_u ? 1 << 2 : 0;

    kernel_ptrs_[kernel_idx]<<<blocks, tpb, 0, stream>>>(
        R_, d_x, d_p, d_idxs_, d_signs_, d_du_dx, d_du_dp,
        d_u == nullptr ? nullptr : d_u_buffer_);
    gpuErrchk(hipPeekAtLastError());

    if (d_u) {
      gpuErrchk(hipcub::DeviceReduce::Sum(d_sum_temp_storage_, sum_storage_bytes_,
                                       d_u_buffer_, d_u, R_, stream));
    }
  }
};

template class ChiralBondRestraint<double>;
template class ChiralBondRestraint<float>;

} // namespace tmd
