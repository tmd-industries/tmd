// Copyright 2025, Forrest York
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
#include "assert.h"
#include "cub_utils.cuh"
#include "energy_accum.hpp"
#include "gpu_utils.cuh"
#include <hipcub/hipcub.hpp>

namespace tmd {

EnergyAccumulator::EnergyAccumulator(const int batches, const int total_size)
    : batches_(batches), max_buffer_size_(total_size), temp_storage_bytes_(0) {
  assert(batches_ >= 1);
  __int128 *dummy_nrg_buffer = nullptr;
  if (batches_ == 1) {
    gpuErrchk(hipcub::DeviceReduce::Sum(nullptr, temp_storage_bytes_,
                                     dummy_nrg_buffer, dummy_nrg_buffer,
                                     max_buffer_size_));
  } else {
    // This is safe as long as the number of batches is accurate to the idxs
    cudaSafeMalloc(&d_reductions_out_, batches * sizeof(*d_reductions_out_));
    cudaSafeMalloc(&d_idxs_unique_, total_size * sizeof(*d_idxs_unique_));
    int *dummy_idxs = nullptr;
    CUBSumOp reduction_op;
    gpuErrchk(hipcub::DeviceReduce::ReduceByKey(
        nullptr, temp_storage_bytes_, dummy_idxs, d_reductions_out_,
        dummy_nrg_buffer, dummy_nrg_buffer, d_reductions_out_, reduction_op,
        max_buffer_size_));
  }

  gpuErrchk(hipMalloc(&d_sum_temp_storage_, temp_storage_bytes_));
}

EnergyAccumulator::~EnergyAccumulator() {
  gpuErrchk(hipFree(d_sum_temp_storage_));
  if (batches_ > 1) {
    gpuErrchk(hipFree(d_reductions_out_));
    gpuErrchk(hipFree(d_idxs_unique_));
  }
};

void EnergyAccumulator::sum_device(const int num_vals, const __int128 *d_nrg_in,
                                   const int *d_system_idxs,
                                   __int128 *d_nrg_out, hipStream_t stream) {

  if (batches_ == 1) {
    gpuErrchk(hipcub::DeviceReduce::Sum(d_sum_temp_storage_, temp_storage_bytes_,
                                     d_nrg_in, d_nrg_out, num_vals, stream));
  } else {
    CUBSumOp reduction_op;
    gpuErrchk(hipcub::DeviceReduce::ReduceByKey(
        d_sum_temp_storage_, temp_storage_bytes_, d_system_idxs, d_idxs_unique_,
        d_nrg_in, d_nrg_out, d_reductions_out_, reduction_op, num_vals,
        stream));
    // TBD: HANDLE THE PERMUTATION IMPLIED BY LOCAL MD
  }
}

} // namespace tmd
