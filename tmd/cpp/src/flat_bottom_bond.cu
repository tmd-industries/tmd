// Copyright 2019-2025, Relay Therapeutics
// Modifications Copyright 2025 Forrest York
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "flat_bottom_bond.hpp"
#include "gpu_utils.cuh"
#include "k_flat_bottom_bond.cuh"
#include "kernel_utils.cuh"
#include "math_utils.cuh"
#include <vector>

namespace tmd {

template <typename RealType>
FlatBottomBond<RealType>::FlatBottomBond(const std::vector<int> &bond_idxs)
    : max_idxs_(bond_idxs.size() / IDXS_DIM), cur_num_idxs_(max_idxs_),
      nrg_accum_(1, cur_num_idxs_),
      kernel_ptrs_({// enumerate over every possible kernel combination
                    // U: Compute U
                    // X: Compute DU_DX
                    // P: Compute DU_DP           U  X  P
                    &k_flat_bottom_bond<RealType, 0, 0, 0>,
                    &k_flat_bottom_bond<RealType, 0, 0, 1>,
                    &k_flat_bottom_bond<RealType, 0, 1, 0>,
                    &k_flat_bottom_bond<RealType, 0, 1, 1>,
                    &k_flat_bottom_bond<RealType, 1, 0, 0>,
                    &k_flat_bottom_bond<RealType, 1, 0, 1>,
                    &k_flat_bottom_bond<RealType, 1, 1, 0>,
                    &k_flat_bottom_bond<RealType, 1, 1, 1>}) {

  // validate bond_idxs: even length, all idxs non-negative, and no self-edges
  if (bond_idxs.size() % IDXS_DIM != 0) {
    throw std::runtime_error("bond_idxs.size() must be exactly " +
                             std::to_string(IDXS_DIM) + "*k!");
  }
  static_assert(IDXS_DIM == 2);
  for (int b = 0; b < cur_num_idxs_; b++) {
    auto src = bond_idxs[b * IDXS_DIM + 0];
    auto dst = bond_idxs[b * IDXS_DIM + 1];
    if (src == dst) {
      throw std::runtime_error("src == dst");
    }

    if ((src < 0) or (dst < 0)) {
      throw std::runtime_error("idxs must be non-negative");
    }
  }

  // copy idxs to device
  cudaSafeMalloc(&d_bond_idxs_, cur_num_idxs_ * 2 * sizeof(*d_bond_idxs_));
  gpuErrchk(hipMemcpy(d_bond_idxs_, &bond_idxs[0],
                       cur_num_idxs_ * 2 * sizeof(*d_bond_idxs_),
                       hipMemcpyHostToDevice));

  cudaSafeMalloc(&d_u_buffer_, cur_num_idxs_ * sizeof(*d_u_buffer_));
};

template <typename RealType> FlatBottomBond<RealType>::~FlatBottomBond() {
  gpuErrchk(hipFree(d_bond_idxs_));
  gpuErrchk(hipFree(d_u_buffer_));
};

template <typename RealType>
void FlatBottomBond<RealType>::execute_device(
    const int batches, const int N, const int P, const RealType *d_x,
    const RealType *d_p, const RealType *d_box, unsigned long long *d_du_dx,
    unsigned long long *d_du_dp, __int128 *d_u, hipStream_t stream) {

  const int num_params_per_bond = 3;
  int expected_P = num_params_per_bond * cur_num_idxs_;

  if (P != expected_P) {
    throw std::runtime_error(
        "FlatBottomBond::execute_device(): expected P == " +
        std::to_string(expected_P) + ", got P=" + std::to_string(P));
  }

  if (cur_num_idxs_ > 0) {
    const int tpb = DEFAULT_THREADS_PER_BLOCK;
    const int blocks = ceil_divide(cur_num_idxs_, tpb);

    int kernel_idx = 0;
    kernel_idx |= d_du_dp ? 1 << 0 : 0;
    kernel_idx |= d_du_dx ? 1 << 1 : 0;
    kernel_idx |= d_u ? 1 << 2 : 0;

    kernel_ptrs_[kernel_idx]<<<blocks, tpb, 0, stream>>>(
        cur_num_idxs_, d_x, d_box, d_p, d_bond_idxs_, d_du_dx, d_du_dp,
        d_u == nullptr ? nullptr : d_u_buffer_);
    gpuErrchk(hipPeekAtLastError());

    if (d_u) {
      // nullptr for the d_system_idxs as batch size is fixed to 1
      nrg_accum_.sum_device(cur_num_idxs_, d_u_buffer_, nullptr, d_u, stream);
    }
  }
};

template <typename RealType>
void FlatBottomBond<RealType>::set_bonds_device(const int num_bonds,
                                                const int *d_bonds,
                                                const hipStream_t stream) {
  if (max_idxs_ < num_bonds) {
    throw std::runtime_error(
        "set_bonds_device(): Max number of bonds " + std::to_string(max_idxs_) +
        " is less than new idxs " + std::to_string(num_bonds));
  }
  gpuErrchk(hipMemcpyAsync(d_bond_idxs_, d_bonds,
                            num_bonds * IDXS_DIM * sizeof(*d_bond_idxs_),
                            hipMemcpyDeviceToDevice, stream));
  cur_num_idxs_ = num_bonds;
}

template class FlatBottomBond<double>;
template class FlatBottomBond<float>;

} // namespace tmd
