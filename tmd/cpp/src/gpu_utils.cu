#include "hip/hip_runtime.h"
// Copyright 2019-2025, Relay Therapeutics
// Modifications Copyright 2025 Forrest York
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "gpu_utils.cuh"

namespace tmd {

int round_up_even(int count) { return count + (count % 2); }

hiprandStatus_t templateCurandNormal(hiprandGenerator_t generator,
                                    float *outputPtr, size_t n, float mean,
                                    float stddev) {
  return hiprandGenerateNormal(generator, outputPtr, n, mean, stddev);
}

hiprandStatus_t templateCurandNormal(hiprandGenerator_t generator,
                                    double *outputPtr, size_t n, double mean,
                                    double stddev) {
  return hiprandGenerateNormalDouble(generator, outputPtr, n, mean, stddev);
}

hiprandStatus_t templateCurandUniform(hiprandGenerator_t generator,
                                     float *outputPtr, size_t n) {
  return hiprandGenerateUniform(generator, outputPtr, n);
}

hiprandStatus_t templateCurandUniform(hiprandGenerator_t generator,
                                     double *outputPtr, size_t n) {
  return hiprandGenerateUniformDouble(generator, outputPtr, n);
}

hipblasStatus_t
templateCublasNorm2(hipblasHandle_t handle, size_t n, float *input_ptr, size_t stride, float *output_ptr) {
    return hipblasSnrm2(handle, n, input_ptr, stride, output_ptr);
}

hipblasStatus_t
templateCublasNorm2(hipblasHandle_t handle, size_t n, double *input_ptr, size_t stride, double *output_ptr) {
    return hipblasDnrm2(handle, n, input_ptr, stride, output_ptr);
}

hipblasStatus_t templateCublasDot(
    hipblasHandle_t handle,
    size_t n,
    float *input_ptr_x,
    size_t x_stride,
    float *input_ptr_y,
    size_t y_stride,
    float *output_ptr) {
    return hipblasSdot(handle, n, input_ptr_x, x_stride, input_ptr_y, y_stride, output_ptr);
}

hipblasStatus_t templateCublasDot(
    hipblasHandle_t handle,
    size_t n,
    double *input_ptr_x,
    size_t x_stride,
    double *input_ptr_y,
    size_t y_stride,
    double *output_ptr) {
    return hipblasDdot(handle, n, input_ptr_x, x_stride, input_ptr_y, y_stride, output_ptr);
}

void __global__ k_initialize_curand_states(const int count, const int seed,
                                           hiprandState_t *__restrict__ states) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  while (idx < count) {
    // Set the sequence to idx to ensure values aren't statistically correlated
    // values
    hiprand_init(seed, idx, 0, &states[idx]);
    idx += gridDim.x * blockDim.x;
  }
}

} // namespace tmd
