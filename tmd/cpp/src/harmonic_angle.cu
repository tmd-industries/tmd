// Copyright 2019-2025, Relay Therapeutics
// Modifications Copyright 2025 Forrest York
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "assert.h"
#include "gpu_utils.cuh"
#include "harmonic_angle.hpp"
#include "k_harmonic_angle.cuh"
#include "kernel_utils.cuh"
#include "math_utils.cuh"
#include <hipcub/hipcub.hpp>
#include <vector>

namespace tmd {

template <typename RealType>
HarmonicAngle<RealType>::HarmonicAngle(
    const std::vector<int> &angle_idxs // [A, 3]
    )
    : max_idxs_(angle_idxs.size() / IDXS_DIM), cur_num_idxs_(max_idxs_),
      sum_storage_bytes_(0),
      kernel_ptrs_({// enumerate over every possible kernel combination
                    // U: Compute U
                    // X: Compute DU_DX
                    // P: Compute DU_DP         U  X  P
                    &k_harmonic_angle<RealType, 0, 0, 0>,
                    &k_harmonic_angle<RealType, 0, 0, 1>,
                    &k_harmonic_angle<RealType, 0, 1, 0>,
                    &k_harmonic_angle<RealType, 0, 1, 1>,
                    &k_harmonic_angle<RealType, 1, 0, 0>,
                    &k_harmonic_angle<RealType, 1, 0, 1>,
                    &k_harmonic_angle<RealType, 1, 1, 0>,
                    &k_harmonic_angle<RealType, 1, 1, 1>}) {

  if (angle_idxs.size() % IDXS_DIM != 0) {
    throw std::runtime_error("angle_idxs.size() must be exactly " +
                             std::to_string(IDXS_DIM) + "*A");
  }
  static_assert(IDXS_DIM == 3);
  for (int a = 0; a < cur_num_idxs_; a++) {
    auto i = angle_idxs[a * IDXS_DIM + 0];
    auto j = angle_idxs[a * IDXS_DIM + 1];
    auto k = angle_idxs[a * IDXS_DIM + 2];
    if (i == j || j == k || i == k) {
      throw std::runtime_error("angle triplets must be unique");
    }
  }

  cudaSafeMalloc(&d_angle_idxs_,
                 cur_num_idxs_ * IDXS_DIM * sizeof(*d_angle_idxs_));
  gpuErrchk(hipMemcpy(d_angle_idxs_, &angle_idxs[0],
                       cur_num_idxs_ * IDXS_DIM * sizeof(*d_angle_idxs_),
                       hipMemcpyHostToDevice));
  cudaSafeMalloc(&d_u_buffer_, cur_num_idxs_ * sizeof(*d_u_buffer_));

  gpuErrchk(hipcub::DeviceReduce::Sum(nullptr, sum_storage_bytes_, d_u_buffer_,
                                   d_u_buffer_, cur_num_idxs_));

  gpuErrchk(hipMalloc(&d_sum_temp_storage_, sum_storage_bytes_));
};

template <typename RealType> HarmonicAngle<RealType>::~HarmonicAngle() {
  gpuErrchk(hipFree(d_angle_idxs_));
  gpuErrchk(hipFree(d_u_buffer_));
  gpuErrchk(hipFree(d_sum_temp_storage_));
};

template <typename RealType>
void HarmonicAngle<RealType>::execute_device(
    const int batches, const int N, const int P, const RealType *d_x,
    const RealType *d_p, const RealType *d_box, unsigned long long *d_du_dx,
    unsigned long long *d_du_dp, __int128 *d_u, hipStream_t stream) {

  assert(batches == 1);
  if (cur_num_idxs_ > 0) {
    const int tpb = DEFAULT_THREADS_PER_BLOCK;
    const int blocks = ceil_divide(cur_num_idxs_, tpb);

    if (P != cur_num_idxs_ * 3) {
      throw std::runtime_error("HarmonicAngle::execute_device(): expected P == "
                               "3*cur_num_idxs_, got P=" +
                               std::to_string(P) + "3*cur_num_idxs_=" +
                               std::to_string(3 * cur_num_idxs_));
    }
    int kernel_idx = 0;
    kernel_idx |= d_du_dp ? 1 << 0 : 0;
    kernel_idx |= d_du_dx ? 1 << 1 : 0;
    kernel_idx |= d_u ? 1 << 2 : 0;

    kernel_ptrs_[kernel_idx]<<<blocks, tpb, 0, stream>>>(
        cur_num_idxs_, d_x, d_box, d_p, d_angle_idxs_, d_du_dx, d_du_dp,
        d_u == nullptr ? nullptr : d_u_buffer_);
    gpuErrchk(hipPeekAtLastError());

    if (d_u) {
      gpuErrchk(hipcub::DeviceReduce::Sum(d_sum_temp_storage_, sum_storage_bytes_,
                                       d_u_buffer_, d_u, cur_num_idxs_,
                                       stream));
    }
  }
}

template <typename RealType>
void HarmonicAngle<RealType>::set_idxs_device(const int num_idxs,
                                              const int *d_new_idxs,
                                              hipStream_t stream) {
  if (max_idxs_ < num_idxs) {
    throw std::runtime_error(
        "set_idxs_device(): Max number of angles " + std::to_string(max_idxs_) +
        " is less than new idxs " + std::to_string(num_idxs));
  }
  gpuErrchk(hipMemcpyAsync(d_angle_idxs_, d_new_idxs,
                            num_idxs * IDXS_DIM * sizeof(*d_angle_idxs_),
                            hipMemcpyDeviceToDevice, stream));
  cur_num_idxs_ = num_idxs;
}

template <typename RealType> int HarmonicAngle<RealType>::get_num_idxs() const {
  return cur_num_idxs_;
}

template <typename RealType> int *HarmonicAngle<RealType>::get_idxs_device() {
  return d_angle_idxs_;
}

template <typename RealType>
std::vector<int> HarmonicAngle<RealType>::get_idxs_host() const {
  return device_array_to_vector<int>(cur_num_idxs_ * IDXS_DIM, d_angle_idxs_);
}

template class HarmonicAngle<double>;
template class HarmonicAngle<float>;

} // namespace tmd
