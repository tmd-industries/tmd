#include "hip/hip_runtime.h"
// Copyright 2019-2025, Relay Therapeutics
// Modifications Copyright 2025 Forrest York
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <algorithm>
#include <numeric>

#include "assert.h"
#include "device_buffer.hpp"
#include "gpu_utils.cuh"
#include "kernels/k_indices.cuh"
#include "kernels/k_neighborlist.cuh"
#include "kernels/k_nonbonded_common.cuh"
#include "neighborlist.hpp"
#include "set_utils.hpp"

namespace tmd {

static int get_idx_total_blocks(const std::vector<int> &idx_counts,
                                const int tile_size) {
  int total_blocks = 0;
  // Need to round up for each set of columns to ensure correct padding
  for (int i = 0; i < idx_counts.size(); i++) {
    total_blocks += ceil_divide(idx_counts[i], tile_size);
  }
  return total_blocks;
}

template <typename RealType>
Neighborlist<RealType>::Neighborlist(const int num_systems, const int N,
                                     const bool compute_upper_triangular)
    : num_systems_(num_systems), max_system_size_(N), N_(N),
      column_idx_counts_(num_systems_), row_idx_counts_(num_systems_),
      compute_upper_triangular_(compute_upper_triangular) {

  if (num_systems == 0) {
    throw std::runtime_error("Neighborlist num_systems must be at least 1");
  }
  if (N == 0) {
    throw std::runtime_error("Neighborlist N must be at least 1");
  }
  // Populate the initial row/column indices
  std::fill(column_idx_counts_.begin(), column_idx_counts_.end(), N_);
  std::fill(row_idx_counts_.begin(), row_idx_counts_.end(), N_);
  const int column_blocks = this->num_column_blocks();
  const int row_blocks = this->num_row_blocks();
  const int Y = this->Y();

  const unsigned long long MAX_TILE_BUFFER =
      num_systems_ * row_blocks * column_blocks;
  const unsigned long long MAX_ATOM_BUFFER =
      num_systems_ * this->max_ixn_count();

  // interaction buffers
  // TBD: Should the ixn count be a single value since use case for this
  // downstream is a single kernel (batched nonbonded kernel)
  cudaSafeMalloc(&d_ixn_count_, num_systems_ * sizeof(*d_ixn_count_));
  cudaSafeMalloc(&d_ixn_tiles_, MAX_TILE_BUFFER * sizeof(*d_ixn_tiles_));
  cudaSafeMalloc(&d_ixn_atoms_, MAX_ATOM_BUFFER * sizeof(*d_ixn_atoms_));
  cudaSafeMalloc(&d_trim_atoms_, num_systems_ * column_blocks * Y * TILE_SIZE *
                                     sizeof(*d_trim_atoms_));

  // bounding box buffers
  cudaSafeMalloc(&d_row_block_bounds_ctr_,
                 num_systems_ * row_blocks * 3 *
                     sizeof(*d_row_block_bounds_ctr_));
  cudaSafeMalloc(&d_row_block_bounds_ext_,
                 num_systems_ * row_blocks * 3 *
                     sizeof(*d_row_block_bounds_ext_));
  cudaSafeMalloc(&d_column_block_bounds_ctr_,
                 num_systems_ * column_blocks * 3 *
                     sizeof(*d_column_block_bounds_ctr_));
  cudaSafeMalloc(&d_column_block_bounds_ext_,
                 num_systems_ * column_blocks * 3 *
                     sizeof(*d_column_block_bounds_ext_));

  // Row and column indice arrays
  cudaSafeMalloc(&d_column_idxs_,
                 num_systems_ * max_system_size_ * sizeof(*d_column_idxs_));
  cudaSafeMalloc(&d_column_system_idxs_, num_systems_ * max_system_size_ *
                                             sizeof(*d_column_system_idxs_));
  cudaSafeMalloc(&d_column_idx_counts_,
                 num_systems_ * sizeof(*d_column_idx_counts_));
  // // Increment by 1 as the first value will be zero
  // cudaSafeMalloc(&d_column_idx_offsets_,
  //                (num_systems_ + 1) * sizeof(*d_column_idx_counts_));
  // gpuErrchk(
  //     hipMemset(d_column_idx_offsets_, 0, (num_systems_ + 1) *
  //     sizeof(*d_column_idx_offsets_)));

  cudaSafeMalloc(&d_row_idxs_,
                 num_systems_ * max_system_size_ * sizeof(*d_row_idxs_));
  cudaSafeMalloc(&d_row_system_idxs_,
                 num_systems_ * max_system_size_ * sizeof(*d_row_system_idxs_));
  cudaSafeMalloc(&d_row_idx_counts_, num_systems_ * sizeof(*d_row_idx_counts_));
  // Increment by 1 as the first value will be zero
  // cudaSafeMalloc(&d_row_idx_offsets_,
  //                (num_systems_ + 1) * sizeof(*d_row_idx_counts_));
  // gpuErrchk(
  //     hipMemset(d_row_idx_offsets_, 0, (num_systems_ + 1) *
  //     sizeof(*d_row_idx_offsets_)));

  // Since d_column_idx_counts_ and d_row_idx_counts_ are the same length can be
  // used for both gpuErrchk(hipcub::DeviceScan::InclusiveSum(
  //     nullptr, temp_sum_bytes_, d_column_idx_counts_,
  //     d_column_idxs_offsets_, num_systems_));

  // cudaSafeMalloc(&d_temp_storage_buffer_, temp_sum_bytes_);

  this->reset_row_idxs();
}

template <typename RealType> Neighborlist<RealType>::~Neighborlist() {
  gpuErrchk(hipFree(d_column_idxs_));
  gpuErrchk(hipFree(d_column_system_idxs_));
  gpuErrchk(hipFree(d_column_idx_counts_));
  gpuErrchk(hipFree(d_row_idxs_));
  gpuErrchk(hipFree(d_row_system_idxs_));
  gpuErrchk(hipFree(d_row_idx_counts_));

  gpuErrchk(hipFree(d_ixn_count_));
  gpuErrchk(hipFree(d_ixn_tiles_));
  gpuErrchk(hipFree(d_ixn_atoms_));
  gpuErrchk(hipFree(d_trim_atoms_));

  gpuErrchk(hipFree(d_row_block_bounds_ctr_));
  gpuErrchk(hipFree(d_row_block_bounds_ext_));
  gpuErrchk(hipFree(d_column_block_bounds_ctr_));
  gpuErrchk(hipFree(d_column_block_bounds_ext_));

  // gpuErrchk(hipFree(d_temp_storage_buffer_));
}

template <typename RealType>
void Neighborlist<RealType>::compute_block_bounds_host(const int N,
                                                       const RealType *h_coords,
                                                       const RealType *h_box,
                                                       RealType *h_bb_ctrs,
                                                       RealType *h_bb_exts) {

  const int D = 3;
  DeviceBuffer<RealType> d_coords(num_systems_ * N * D);
  DeviceBuffer<RealType> d_box(num_systems_ * D * D);

  d_coords.copy_from(h_coords);
  d_box.copy_from(h_box);

  this->compute_block_bounds_device(N, D, d_coords.data, d_box.data,
                                    static_cast<hipStream_t>(0));
  gpuErrchk(hipDeviceSynchronize());

  gpuErrchk(hipMemcpy(h_bb_ctrs, d_column_block_bounds_ctr_,
                       this->num_column_blocks() * 3 *
                           sizeof(*d_column_block_bounds_ctr_),
                       hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(h_bb_exts, d_column_block_bounds_ext_,
                       this->num_column_blocks() * 3 *
                           sizeof(*d_column_block_bounds_ext_),
                       hipMemcpyDeviceToHost));
}

// Return the number of tiles that interact
template <typename RealType>
unsigned int Neighborlist<RealType>::num_tile_ixns() {
  unsigned int h_ixn_count;
  gpuErrchk(hipMemcpy(&h_ixn_count, d_ixn_count_, 1 * sizeof(*d_ixn_count_),
                       hipMemcpyDeviceToHost));
  return h_ixn_count;
}

template <typename RealType>
std::vector<std::vector<int>> Neighborlist<RealType>::get_nblist_host(
    int N, const RealType *h_coords, const RealType *h_box,
    const RealType cutoff, const RealType padding) {

  if (N != N_) {
    throw std::runtime_error("N != N_");
  }

  DeviceBuffer<RealType> d_coords(N * 3);
  DeviceBuffer<RealType> d_box(3 * 3);
  d_coords.copy_from(h_coords);
  d_box.copy_from(h_box);

  this->build_nblist_device(N, d_coords.data, d_box.data, cutoff, padding,
                            static_cast<hipStream_t>(0));

  gpuErrchk(hipDeviceSynchronize());
  const int column_blocks = this->num_column_blocks();
  const int row_blocks = this->num_row_blocks();

  unsigned long long MAX_TILE_BUFFER = row_blocks * column_blocks;
  unsigned long long MAX_ATOM_BUFFER = this->max_ixn_count();

  unsigned int h_ixn_count;
  gpuErrchk(hipMemcpy(&h_ixn_count, d_ixn_count_, 1 * sizeof(*d_ixn_count_),
                       hipMemcpyDeviceToHost));
  std::vector<int> h_ixn_tiles(MAX_TILE_BUFFER);
  std::vector<unsigned int> h_ixn_atoms(MAX_ATOM_BUFFER);
  gpuErrchk(hipMemcpy(&h_ixn_tiles[0], d_ixn_tiles_,
                       MAX_TILE_BUFFER * sizeof(*d_ixn_tiles_),
                       hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(&h_ixn_atoms[0], d_ixn_atoms_,
                       MAX_ATOM_BUFFER * sizeof(*d_ixn_atoms_),
                       hipMemcpyDeviceToHost));

  std::vector<std::vector<int>> ixn_list(row_blocks, std::vector<int>());
  for (int i = 0; i < h_ixn_count; i++) {
    int tile_idx = h_ixn_tiles[i];
    for (int j = 0; j < TILE_SIZE; j++) {
      int atom_j_idx = h_ixn_atoms[i * TILE_SIZE + j];
      if (atom_j_idx < N) {
        ixn_list[tile_idx].push_back(atom_j_idx);
      }
    }
  }

  return ixn_list;
}

template <typename RealType>
void Neighborlist<RealType>::build_nblist_device(
    const int N, const RealType *d_coords, const RealType *d_box,
    const RealType cutoff, const RealType padding, const hipStream_t stream) {

  const int D = 3;
  this->compute_block_bounds_device(N, D, d_coords, d_box, stream);
  const int tpb = TILE_SIZE;
  const int row_blocks = this->num_row_blocks();
  const int Y = this->Y();

  dim3 dimGrid(row_blocks, Y, 1); // block x, y, z dims

  // (ytz): TBD shared memory, stream
  if (this->compute_upper_triangular()) {
    k_find_blocks_with_ixns<RealType, true><<<dimGrid, tpb, 0, stream>>>(
        num_systems_, N_, d_column_idx_counts_, d_row_idx_counts_,
        d_column_idxs_, d_row_idxs_, d_column_block_bounds_ctr_,
        d_column_block_bounds_ext_, d_column_block_bounds_ctr_,
        d_column_block_bounds_ext_, d_coords, d_box, d_ixn_count_, d_ixn_tiles_,
        d_ixn_atoms_, d_trim_atoms_, cutoff, padding);
  } else {
    k_find_blocks_with_ixns<RealType, false><<<dimGrid, tpb, 0, stream>>>(
        num_systems_, N_, d_column_idx_counts_, d_row_idx_counts_,
        d_column_idxs_, d_row_idxs_, d_column_block_bounds_ctr_,
        d_column_block_bounds_ext_, d_row_block_bounds_ctr_,
        d_row_block_bounds_ext_, d_coords, d_box, d_ixn_count_, d_ixn_tiles_,
        d_ixn_atoms_, d_trim_atoms_, cutoff, padding);
  }

  gpuErrchk(hipPeekAtLastError());
  k_compact_trim_atoms<<<row_blocks, tpb, 0, stream>>>(
      N_, Y, d_trim_atoms_, d_ixn_count_, d_ixn_tiles_, d_ixn_atoms_);

  gpuErrchk(hipPeekAtLastError());
}

template <typename RealType>
void Neighborlist<RealType>::compute_block_bounds_device(
    const int N,              // Number of atoms
    const int D,              // Box dimensions
    const RealType *d_coords, // [N*3]
    const RealType *d_box,    // [D*3]
    const hipStream_t stream) {

  if (D != 3) {
    throw std::runtime_error("D != 3");
  }

  const int tpb = DEFAULT_THREADS_PER_BLOCK;
  const int max_col_idxs =
      *std::max_element(column_idx_counts_.begin(), column_idx_counts_.end());
  dim3 col_dim_grid(ceil_divide(max_col_idxs, tpb), num_systems_, 1);
  printf("Max col idxs %d\n", max_col_idxs);
  k_find_block_bounds<RealType><<<col_dim_grid, tpb, 0, stream>>>(
      num_systems_, N_, d_column_idx_counts_, d_column_idxs_, d_coords, d_box,
      d_column_block_bounds_ctr_, d_column_block_bounds_ext_, d_ixn_count_);
  gpuErrchk(hipPeekAtLastError());
  // In the case of upper triangle of the matrix, the column and row indices are
  // the same, so only compute block ixns for both when they are different

  // We have three possible scenarios:
  // If we compute upper triangular:
  // - we're in the all-pairs case, row_idxs are equal to col_idxs.
  // - we're in the overlapping rectangular case, row_idxs are a subset of
  // col_idxs. If we do not compute upper triangular:
  // - we're in the disjoint rectangular case, row_idxs need to be processed as
  // well.
  if (!this->compute_upper_triangular()) {
    const int max_row_idxs =
        *std::max_element(row_idx_counts_.begin(), row_idx_counts_.end());
    dim3 row_dim_grid(ceil_divide(max_row_idxs, tpb), num_systems_, 1);

    k_find_block_bounds<RealType><<<row_dim_grid, tpb, 0, stream>>>(
        num_systems_, N_, d_row_idx_counts_, d_row_idxs_, d_coords, d_box,
        d_row_block_bounds_ctr_, d_row_block_bounds_ext_, d_ixn_count_);
    gpuErrchk(hipPeekAtLastError());
  }
};

template <typename RealType>
void Neighborlist<RealType>::set_compute_upper_triangular(bool val) {
  compute_upper_triangular_ = val;
}

template <typename RealType>
void Neighborlist<RealType>::set_row_idxs(std::vector<unsigned int> &row_idxs) {
  std::set<unsigned int> unique_row_idxs(row_idxs.begin(), row_idxs.end());
  std::vector<unsigned int> col_idxs =
      get_indices_difference<unsigned int>(N_, unique_row_idxs);
  this->set_row_idxs_and_col_idxs(row_idxs, col_idxs);
}

template <typename RealType>
void Neighborlist<RealType>::set_row_idxs_and_col_idxs(
    std::vector<unsigned int> &row_idxs, std::vector<unsigned int> &col_idxs) {
  if (row_idxs.size() == 0) {
    throw std::runtime_error("idxs can't be empty");
  }
  std::set<unsigned int> unique_row_idxs(row_idxs.begin(), row_idxs.end());
  if (unique_row_idxs.size() != row_idxs.size()) {
    throw std::runtime_error("atom indices must be unique");
  }
  if (row_idxs.size() >= N_) {
    throw std::runtime_error("number of idxs must be less than N");
  }
  if (*std::max_element(row_idxs.begin(), row_idxs.end()) >= N_) {
    throw std::runtime_error("indices values must be less than N");
  }
  if (col_idxs.size() == 0) {
    throw std::runtime_error("idxs can't be empty");
  }
  std::set<unsigned int> unique_col_idxs(col_idxs.begin(), col_idxs.end());
  if (unique_col_idxs.size() != col_idxs.size()) {
    throw std::runtime_error("atom indices must be unique");
  }
  if (col_idxs.size() > N_) {
    throw std::runtime_error("number of col idxs must be <= N");
  }
  if (*std::max_element(col_idxs.begin(), col_idxs.end()) >= N_) {
    throw std::runtime_error("indices values must be less than N");
  }

  const size_t row_count = row_idxs.size();
  const size_t col_count = col_idxs.size();

  DeviceBuffer<unsigned int> row_idx_buffer(row_count);
  DeviceBuffer<unsigned int> col_idx_buffer(col_count);

  row_idx_buffer.copy_from(&row_idxs[0]);
  col_idx_buffer.copy_from(&col_idxs[0]);
  // note, this passes in col/row as opposed to row/col
  this->set_idxs_device(col_count, row_count, col_idx_buffer.data,
                        row_idx_buffer.data, static_cast<hipStream_t>(0));
  gpuErrchk(hipDeviceSynchronize());
}

template <typename RealType> void Neighborlist<RealType>::reset_row_idxs() {
  const hipStream_t stream = static_cast<hipStream_t>(0);
  this->reset_row_idxs_device(stream);
  gpuErrchk(hipStreamSynchronize(stream));
}

template <typename RealType>
void Neighborlist<RealType>::reset_row_idxs_device(const hipStream_t stream) {
  const int tpb = DEFAULT_THREADS_PER_BLOCK;

  dim3 dimGrid(ceil_divide(N_, tpb), num_systems_, 1); // block x, y, z dims
  // Fill the indices with the 0 to N-1 indices, indicating 'normal'
  // neighborlist operation
  // printf("Num Systems %d\n", num_systems_);
  k_segment_arange<unsigned int>
      <<<dimGrid, tpb, 0, stream>>>(num_systems_, N_, d_column_idxs_);
  gpuErrchk(hipPeekAtLastError());
  k_segment_arange<unsigned int>
      <<<dimGrid, tpb, 0, stream>>>(num_systems_, N_, d_row_idxs_);
  gpuErrchk(hipPeekAtLastError());

  k_reset_system_idxs<<<dimGrid, tpb, 0, stream>>>(
      num_systems_, N_, d_column_system_idxs_, d_row_system_idxs_);
  gpuErrchk(hipPeekAtLastError());

  k_fill<<<ceil_divide(num_systems_, tpb), tpb, 0, stream>>>(
      num_systems_, d_column_idx_counts_, static_cast<unsigned int>(N_));
  gpuErrchk(hipPeekAtLastError());
  k_fill<<<ceil_divide(num_systems_, tpb), tpb, 0, stream>>>(
      num_systems_, d_row_idx_counts_, static_cast<unsigned int>(N_));
  gpuErrchk(hipPeekAtLastError());

  // // Setup the offsets
  // // Offset output buffers by 1 since the first value is always zero
  // gpuErrchk(hipcub::DeviceScan::InclusiveSum(
  //     d_temp_storage_buffer_, temp_sum_bytes_, d_column_idx_counts_,
  //     d_column_idxs_offsets_ + 1, num_systems_));

  // gpuErrchk(hipcub::DeviceScan::InclusiveSum(
  //     d_temp_storage_buffer_, temp_sum_bytes_, d_row_idx_counts_,
  //     d_row_idxs_offsets_ + 1, num_systems_));
}

template <typename RealType>
void Neighborlist<RealType>::resize(const int size) {
  const hipStream_t stream = static_cast<hipStream_t>(0);
  this->resize_device(size, stream);
  gpuErrchk(hipStreamSynchronize(stream));
}

// Resize the Neighborlist to function on a different size. Note that this only
// allows finding interactions on a smaller set of the system, will not increase
// the size of the underlying buffers.
template <typename RealType>
void Neighborlist<RealType>::resize_device(const int size,
                                           const hipStream_t stream) {
  if (size <= 0) {
    throw std::runtime_error("size is must be at least 1");
  }
  if (size > max_system_size_) {
    throw std::runtime_error(
        "size is greater than max size: " + std::to_string(size) + " > " +
        std::to_string(max_system_size_));
  }
  this->N_ = size;
  this->reset_row_idxs_device(stream);
}

// set_idxs_device is for use when idxs exist on the GPU already and are used as
// the new idxs to compute the neighborlist on.
template <typename RealType>
void Neighborlist<RealType>::set_idxs_device(const int NC, const int NR,
                                             unsigned int *d_in_column_idxs,
                                             unsigned int *d_in_row_idxs,
                                             const hipStream_t stream) {
  // TBD: Change the input to be vectors
  assert(num_systems_ == 1);
  if (NC > N_) {
    throw std::runtime_error("NC > N_ ");
  }
  if (NR > N_) {
    throw std::runtime_error("NR > N_ ");
  }
  if (NC == 0 || NR == 0) {
    throw std::runtime_error(
        "Number of column and row indices must be non-zero");
  }
  const size_t tpb = DEFAULT_THREADS_PER_BLOCK;

  // initialize both to N_
  k_initialize_array<unsigned int>
      <<<ceil_divide(max_system_size_, tpb), tpb, 0, stream>>>(
          max_system_size_, d_column_idxs_, N_);
  gpuErrchk(hipPeekAtLastError());
  k_initialize_array<unsigned int>
      <<<ceil_divide(max_system_size_, tpb), tpb, 0, stream>>>(max_system_size_,
                                                               d_row_idxs_, N_);
  gpuErrchk(hipPeekAtLastError());

  // The indices must already be on the GPU and are copied into the neighborlist
  // buffers.
  gpuErrchk(hipMemcpyAsync(d_column_idxs_, d_in_column_idxs,
                            num_systems_ * NC * sizeof(*d_column_idxs_),
                            hipMemcpyDeviceToDevice, stream));
  gpuErrchk(hipMemcpyAsync(d_row_idxs_, d_in_row_idxs,
                            num_systems_ * NR * sizeof(*d_row_idxs_),
                            hipMemcpyDeviceToDevice, stream));

  // Update the row and column counts
  // TBD: Write this to the GPU
  std::fill(row_idx_counts_.begin(), row_idx_counts_.end(), NR);
  std::fill(column_idx_counts_.begin(), column_idx_counts_.end(), NC);

  // TBD: Avoid the memcpy from host to device (should be easily done, but then
  // the host copies need to be updated)
  gpuErrchk(hipMemcpyAsync(d_column_idx_counts_, &column_idx_counts_[0],
                            num_systems_ * sizeof(*d_column_idx_counts_),
                            hipMemcpyHostToDevice, stream));
  gpuErrchk(hipMemcpyAsync(d_row_idx_counts_, &row_idx_counts_[0],
                            num_systems_ * sizeof(*d_row_idx_counts_),
                            hipMemcpyHostToDevice, stream));

  // TBD: Decide jank of where num_systems_ gets applied
  const unsigned long long MAX_ATOM_BUFFER =
      num_systems_ * this->max_ixn_count();
  // Clear the atom ixns, to avoid reuse
  // Set to max value, ie greater than N. Note that Memset is on bytes, which is
  // why it is UCHAR_MAX
  gpuErrchk(hipMemsetAsync(d_ixn_atoms_, UCHAR_MAX,
                            MAX_ATOM_BUFFER * sizeof(*d_ixn_atoms_), stream));
}

template <typename RealType>
bool Neighborlist<RealType>::compute_upper_triangular() const {
  return compute_upper_triangular_;
};

template <typename RealType>
int Neighborlist<RealType>::num_column_blocks() const {
  return get_idx_total_blocks(column_idx_counts_, TILE_SIZE);
  int total_col_blocks = 0;
  // Need to round up for each set of columns to ensure correct padding
  for (int i = 0; i < column_idx_counts_.size(); i++) {
    total_col_blocks += ceil_divide(column_idx_counts_[i], TILE_SIZE);
  }
  return total_col_blocks;
};

template <typename RealType> int Neighborlist<RealType>::Y() const {
  const int max_column_blocks =
      *std::max_element(column_idx_counts_.begin(), column_idx_counts_.end());
  // Doesn't scale with the number of systems
  return ceil_divide(max_column_blocks, WARP_SIZE);
};

template <typename RealType>
int Neighborlist<RealType>::num_row_blocks() const {
  return get_idx_total_blocks(row_idx_counts_, TILE_SIZE);
}

template <typename RealType>
int Neighborlist<RealType>::total_column_idxs() const {
  return std::reduce(column_idx_counts_.begin(), column_idx_counts_.end());
}

template <typename RealType>
int Neighborlist<RealType>::total_row_idxs() const {
  return std::reduce(row_idx_counts_.begin(), row_idx_counts_.end());
}

// max_ixn_count determines the number of tile-atom interaction counts. For each
// tile that interacts with another it can have TILE_SIZE tile-atom
// interactions. Note that d_ixn_count_ is only the number of tile-tile
// interactions, and differs by a factor of TILE_SIZE
template <typename RealType> int Neighborlist<RealType>::max_ixn_count() const {
  // The maximum number of tile-atom interactions, equal to # tile-tile
  // interactions multiplied by TILE_SIZE (typically 32). Use the maximum value
  // of N to compute the size int of the upper triangular matrix to support any
  // set of row indices.
  const int n_blocks = ceil_divide(max_system_size_, TILE_SIZE);
  int max_tile_tile_interactions = (n_blocks * (n_blocks + 1)) / 2;
  // Each tile-tile interaction can have TILE_SIZE tile-atom interactions
  return max_tile_tile_interactions * TILE_SIZE;
}

template class Neighborlist<double>;
template class Neighborlist<float>;

} // namespace tmd
