#include "hip/hip_runtime.h"
// Copyright 2019-2025, Relay Therapeutics
// Modifications Copyright 2025 Forrest York
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <algorithm>
#include <vector>

#include "device_buffer.hpp"
#include "gpu_utils.cuh"
#include "kernels/k_indices.cuh"
#include "kernels/k_neighborlist.cuh"
#include "kernels/k_nonbonded_common.cuh"
#include "neighborlist.hpp"
#include "set_utils.hpp"

namespace tmd {

template <typename RealType>
Neighborlist<RealType>::Neighborlist(const num_systems, const int N, const bool compute_upper_triangular)
    : num_systems_(num_systems), max_size_(N), N_(N), NC_(N), NR_(N),
      compute_upper_triangular_(compute_upper_triangular) {

  if (num_systems == 0) {
    throw std::runtime_error("Neighborlist num_systems must be at least 1");
  }
  if (N == 0) {
    throw std::runtime_error("Neighborlist N must be at least 1");
  }
  const int column_blocks = this->num_column_blocks();
  const int row_blocks = this->num_row_blocks();
  const int Y = this->Y();

  const unsigned long long MAX_TILE_BUFFER = num_systems_ * row_blocks * column_blocks;
  const unsigned long long MAX_ATOM_BUFFER = num_systems_ *  this->max_ixn_count();

  // interaction buffers
  cudaSafeMalloc(&d_ixn_count_, num_systems_ * sizeof(*d_ixn_count_));
  cudaSafeMalloc(&d_ixn_tiles_, MAX_TILE_BUFFER * sizeof(*d_ixn_tiles_));
  cudaSafeMalloc(&d_ixn_atoms_, MAX_ATOM_BUFFER * sizeof(*d_ixn_atoms_));
  cudaSafeMalloc(&d_trim_atoms_,
                 num_systems_ * column_blocks * Y * TILE_SIZE * sizeof(*d_trim_atoms_));

  // bounding box buffers
  cudaSafeMalloc(&d_row_block_bounds_ctr_,
                 num_systems_ * row_blocks * 3 * sizeof(*d_row_block_bounds_ctr_));
  cudaSafeMalloc(&d_row_block_bounds_ext_,
                 num_systems_ * row_blocks * 3 * sizeof(*d_row_block_bounds_ext_));
  cudaSafeMalloc(&d_column_block_bounds_ctr_,
                 num_systems_ * column_blocks * 3 * sizeof(*d_column_block_bounds_ctr_));
  cudaSafeMalloc(&d_column_block_bounds_ext_,
                 num_systems_ * column_blocks * 3 * sizeof(*d_column_block_bounds_ext_));

  // Row and column indice arrays
  cudaSafeMalloc(&d_column_idxs_, num_systems_* max_size_ * sizeof(*d_column_idxs_));
  cudaSafeMalloc(&d_row_idxs_, num_systems_* max_size_ * sizeof(*d_row_idxs_));

  this->reset_row_idxs();
}

template <typename RealType> Neighborlist<RealType>::~Neighborlist() {
  gpuErrchk(hipFree(d_column_idxs_));
  gpuErrchk(hipFree(d_row_idxs_));

  gpuErrchk(hipFree(d_ixn_count_));
  gpuErrchk(hipFree(d_ixn_tiles_));
  gpuErrchk(hipFree(d_ixn_atoms_));
  gpuErrchk(hipFree(d_trim_atoms_));

  gpuErrchk(hipFree(d_row_block_bounds_ctr_));
  gpuErrchk(hipFree(d_row_block_bounds_ext_));
  gpuErrchk(hipFree(d_column_block_bounds_ctr_));
  gpuErrchk(hipFree(d_column_block_bounds_ext_));
}

template <typename RealType>
void Neighborlist<RealType>::compute_block_bounds_host(const int N,
                                                       const RealType *h_coords,
                                                       const RealType *h_box,
                                                       RealType *h_bb_ctrs,
                                                       RealType *h_bb_exts) {

  const int D = 3;
  DeviceBuffer<RealType> d_coords(num_systems_, N * D);
  DeviceBuffer<RealType> d_box(num_systems_, D * D);

  d_coords.copy_from(h_coords);
  d_box.copy_from(h_box);

  this->compute_block_bounds_device(N, D, d_coords.data, d_box.data,
                                    static_cast<hipStream_t>(0));
  gpuErrchk(hipDeviceSynchronize());

  gpuErrchk(hipMemcpy(h_bb_ctrs, d_column_block_bounds_ctr_,
                       this->num_column_blocks() * 3 *
                           sizeof(*d_column_block_bounds_ctr_),
                       hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(h_bb_exts, d_column_block_bounds_ext_,
                       this->num_column_blocks() * 3 *
                           sizeof(*d_column_block_bounds_ext_),
                       hipMemcpyDeviceToHost));
}

// Return the number of tiles that interact
template <typename RealType>
unsigned int Neighborlist<RealType>::num_tile_ixns() {
  unsigned int h_ixn_count;
  gpuErrchk(hipMemcpy(&h_ixn_count, d_ixn_count_, 1 * sizeof(*d_ixn_count_),
                       hipMemcpyDeviceToHost));
  return h_ixn_count;
}

template <typename RealType>
std::vector<std::vector<int>> Neighborlist<RealType>::get_nblist_host(
    int N, const RealType *h_coords, const RealType *h_box,
    const RealType cutoff, const RealType padding) {

  if (N != N_) {
    throw std::runtime_error("N != N_");
  }

  DeviceBuffer<RealType> d_coords(N * 3);
  DeviceBuffer<RealType> d_box(3 * 3);
  d_coords.copy_from(h_coords);
  d_box.copy_from(h_box);

  this->build_nblist_device(N, d_coords.data, d_box.data, cutoff, padding,
                            static_cast<hipStream_t>(0));

  gpuErrchk(hipDeviceSynchronize());
  const int column_blocks = this->num_column_blocks();
  const int row_blocks = this->num_row_blocks();

  unsigned long long MAX_TILE_BUFFER = row_blocks * column_blocks;
  unsigned long long MAX_ATOM_BUFFER = this->max_ixn_count();

  unsigned int h_ixn_count;
  gpuErrchk(hipMemcpy(&h_ixn_count, d_ixn_count_, 1 * sizeof(*d_ixn_count_),
                       hipMemcpyDeviceToHost));
  std::vector<int> h_ixn_tiles(MAX_TILE_BUFFER);
  std::vector<unsigned int> h_ixn_atoms(MAX_ATOM_BUFFER);
  gpuErrchk(hipMemcpy(&h_ixn_tiles[0], d_ixn_tiles_,
                       MAX_TILE_BUFFER * sizeof(*d_ixn_tiles_),
                       hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(&h_ixn_atoms[0], d_ixn_atoms_,
                       MAX_ATOM_BUFFER * sizeof(*d_ixn_atoms_),
                       hipMemcpyDeviceToHost));

  std::vector<std::vector<int>> ixn_list(row_blocks, std::vector<int>());
  for (int i = 0; i < h_ixn_count; i++) {
    int tile_idx = h_ixn_tiles[i];
    for (int j = 0; j < TILE_SIZE; j++) {
      int atom_j_idx = h_ixn_atoms[i * TILE_SIZE + j];
      if (atom_j_idx < N) {
        ixn_list[tile_idx].push_back(atom_j_idx);
      }
    }
  }

  return ixn_list;
}

template <typename RealType>
void Neighborlist<RealType>::build_nblist_device(
    const int N, const RealType *d_coords, const RealType *d_box,
    const RealType cutoff, const RealType padding, const hipStream_t stream) {

  const int D = 3;
  this->compute_block_bounds_device(N, D, d_coords, d_box, stream);
  const int tpb = TILE_SIZE;
  const int row_blocks = this->num_row_blocks();
  const int Y = this->Y();

  dim3 dimGrid(row_blocks, Y, 1); // block x, y, z dims

  // (ytz): TBD shared memory, stream
  if (this->compute_upper_triangular()) {
    k_find_blocks_with_ixns<RealType, true><<<dimGrid, tpb, 0, stream>>>(
        N_, NC_, NR_, d_column_idxs_, d_row_idxs_, d_column_block_bounds_ctr_,
        d_column_block_bounds_ext_, d_column_block_bounds_ctr_,
        d_column_block_bounds_ext_, d_coords, d_box, d_ixn_count_, d_ixn_tiles_,
        d_ixn_atoms_, d_trim_atoms_, cutoff, padding);
  } else {
    k_find_blocks_with_ixns<RealType, false><<<dimGrid, tpb, 0, stream>>>(
        N_, NC_, NR_, d_column_idxs_, d_row_idxs_, d_column_block_bounds_ctr_,
        d_column_block_bounds_ext_, d_row_block_bounds_ctr_,
        d_row_block_bounds_ext_, d_coords, d_box, d_ixn_count_, d_ixn_tiles_,
        d_ixn_atoms_, d_trim_atoms_, cutoff, padding);
  }

  gpuErrchk(hipPeekAtLastError());
  k_compact_trim_atoms<<<row_blocks, tpb, 0, stream>>>(
      N_, Y, d_trim_atoms_, d_ixn_count_, d_ixn_tiles_, d_ixn_atoms_);

  gpuErrchk(hipPeekAtLastError());
}

template <typename RealType>
void Neighborlist<RealType>::compute_block_bounds_device(
    const int N,              // Number of atoms
    const int D,              // Box dimensions
    const RealType *d_coords, // [N*3]
    const RealType *d_box,    // [D*3]
    const hipStream_t stream) {

  if (D != 3) {
    throw std::runtime_error("D != 3");
  }

  const int tpb = DEFAULT_THREADS_PER_BLOCK;

  k_find_block_bounds<RealType><<<ceil_divide(NC_, tpb), tpb, 0, stream>>>(
      this->num_column_blocks(), NC_, d_column_idxs_, d_coords, d_box,
      d_column_block_bounds_ctr_, d_column_block_bounds_ext_, d_ixn_count_);
  gpuErrchk(hipPeekAtLastError());
  // In the case of upper triangle of the matrix, the column and row indices are
  // the same, so only compute block ixns for both when they are different

  // We have three possible scenarios:
  // If we compute upper triangular:
  // - we're in the all-pairs case, row_idxs are equal to col_idxs.
  // - we're in the overlapping rectangular case, row_idxs are a subset of
  // col_idxs. If we do not compute upper triangular:
  // - we're in the disjoint rectangular case, row_idxs need to be processed as
  // well.
  if (!this->compute_upper_triangular()) {
    k_find_block_bounds<RealType><<<ceil_divide(NR_, tpb), tpb, 0, stream>>>(
        this->num_row_blocks(), NR_, d_row_idxs_, d_coords, d_box,
        d_row_block_bounds_ctr_, d_row_block_bounds_ext_, d_ixn_count_);
    gpuErrchk(hipPeekAtLastError());
  }
};

template <typename RealType>
void Neighborlist<RealType>::set_compute_upper_triangular(bool val) {
  compute_upper_triangular_ = val;
}

template <typename RealType>
void Neighborlist<RealType>::set_row_idxs(std::vector<unsigned int> &row_idxs) {
  std::set<unsigned int> unique_row_idxs(row_idxs.begin(), row_idxs.end());
  std::vector<unsigned int> col_idxs =
      get_indices_difference<unsigned int>(N_, unique_row_idxs);
  this->set_row_idxs_and_col_idxs(row_idxs, col_idxs);
}

template <typename RealType>
void Neighborlist<RealType>::set_row_idxs_and_col_idxs(
    std::vector<unsigned int> &row_idxs, std::vector<unsigned int> &col_idxs) {
  if (row_idxs.size() == 0) {
    throw std::runtime_error("idxs can't be empty");
  }
  std::set<unsigned int> unique_row_idxs(row_idxs.begin(), row_idxs.end());
  if (unique_row_idxs.size() != row_idxs.size()) {
    throw std::runtime_error("atom indices must be unique");
  }
  if (row_idxs.size() >= N_) {
    throw std::runtime_error("number of idxs must be less than N");
  }
  if (*std::max_element(row_idxs.begin(), row_idxs.end()) >= N_) {
    throw std::runtime_error("indices values must be less than N");
  }
  if (col_idxs.size() == 0) {
    throw std::runtime_error("idxs can't be empty");
  }
  std::set<unsigned int> unique_col_idxs(col_idxs.begin(), col_idxs.end());
  if (unique_col_idxs.size() != col_idxs.size()) {
    throw std::runtime_error("atom indices must be unique");
  }
  if (col_idxs.size() > N_) {
    throw std::runtime_error("number of col idxs must be <= N");
  }
  if (*std::max_element(col_idxs.begin(), col_idxs.end()) >= N_) {
    throw std::runtime_error("indices values must be less than N");
  }

  const size_t row_count = row_idxs.size();
  const size_t col_count = col_idxs.size();

  DeviceBuffer<unsigned int> row_idx_buffer(row_count);
  DeviceBuffer<unsigned int> col_idx_buffer(col_count);

  row_idx_buffer.copy_from(&row_idxs[0]);
  col_idx_buffer.copy_from(&col_idxs[0]);
  // note, this passes in col/row as opposed to row/col
  this->set_idxs_device(col_count, row_count, col_idx_buffer.data,
                        row_idx_buffer.data, static_cast<hipStream_t>(0));
  gpuErrchk(hipDeviceSynchronize());
}

template <typename RealType> void Neighborlist<RealType>::reset_row_idxs() {
  const hipStream_t stream = static_cast<hipStream_t>(0);
  this->reset_row_idxs_device(stream);
  gpuErrchk(hipStreamSynchronize(stream));
}

template <typename RealType>
void Neighborlist<RealType>::reset_row_idxs_device(const hipStream_t stream) {
  const int tpb = DEFAULT_THREADS_PER_BLOCK;
  const int blocks = ceil_divide(N_, tpb);
  // Fill the indices with the 0 to N-1 indices, indicating 'normal'
  // neighborlist operation
  k_arange<<<blocks, tpb, 0, stream>>>(N_, d_column_idxs_);
  gpuErrchk(hipPeekAtLastError());
  k_arange<<<blocks, tpb, 0, stream>>>(N_, d_row_idxs_);
  gpuErrchk(hipPeekAtLastError());
  this->NR_ = N_;
  this->NC_ = N_;
}

template <typename RealType>
void Neighborlist<RealType>::resize(const int size) {
  const hipStream_t stream = static_cast<hipStream_t>(0);
  this->resize_device(size, stream);
  gpuErrchk(hipStreamSynchronize(stream));
}

// Resize the Neighborlist to function on a different size. Note that this only
// allows finding interactions on a smaller set of the system, will not increase
// the size of the underlying buffers.
template <typename RealType>
void Neighborlist<RealType>::resize_device(const int size,
                                           const hipStream_t stream) {
  if (size <= 0) {
    throw std::runtime_error("size is must be at least 1");
  }
  if (size > max_size_) {
    throw std::runtime_error(
        "size is greater than max size: " + std::to_string(size) + " > " +
        std::to_string(max_size_));
  }
  this->N_ = size;
  this->reset_row_idxs_device(stream);
}

// set_idxs_device is for use when idxs exist on the GPU already and are used as
// the new idxs to compute the neighborlist on.
template <typename RealType>
void Neighborlist<RealType>::set_idxs_device(const int NC, const int NR,
                                             unsigned int *d_in_column_idxs,
                                             unsigned int *d_in_row_idxs,
                                             const hipStream_t stream) {
  if (NC > N_) {
    throw std::runtime_error("NC > N_ ");
  }
  if (NR > N_) {
    throw std::runtime_error("NR > N_ ");
  }
  if (NC == 0 || NR == 0) {
    throw std::runtime_error(
        "Number of column and row indices must be non-zero");
  }
  const size_t tpb = DEFAULT_THREADS_PER_BLOCK;

  // initialize both to N_
  k_initialize_array<unsigned int>
      <<<ceil_divide(max_size_, tpb), tpb, 0, stream>>>(max_size_,
                                                        d_column_idxs_, N_);
  gpuErrchk(hipPeekAtLastError());
  k_initialize_array<unsigned int>
      <<<ceil_divide(max_size_, tpb), tpb, 0, stream>>>(max_size_, d_row_idxs_,
                                                        N_);
  gpuErrchk(hipPeekAtLastError());

  // The indices must already be on the GPU and are copied into the neighborlist
  // buffers.
  gpuErrchk(hipMemcpyAsync(d_column_idxs_, d_in_column_idxs,
                            NC * sizeof(*d_column_idxs_),
                            hipMemcpyDeviceToDevice, stream));
  gpuErrchk(hipMemcpyAsync(d_row_idxs_, d_in_row_idxs,
                            NR * sizeof(*d_row_idxs_), hipMemcpyDeviceToDevice,
                            stream));

  // Update the row and column counts
  this->NR_ = NR;
  this->NC_ = NC;

  const unsigned long long MAX_ATOM_BUFFER = this->max_ixn_count();
  // Clear the atom ixns, to avoid reuse
  // Set to max value, ie greater than N. Note that Memset is on bytes, which is
  // why it is UCHAR_MAX
  gpuErrchk(hipMemsetAsync(d_ixn_atoms_, UCHAR_MAX,
                            MAX_ATOM_BUFFER * sizeof(*d_ixn_atoms_), stream));
}

template <typename RealType>
bool Neighborlist<RealType>::compute_upper_triangular() const {
  return compute_upper_triangular_;
};

template <typename RealType>
int Neighborlist<RealType>::num_column_blocks() const {
  return ceil_divide(NC_, TILE_SIZE);
};

template <typename RealType> int Neighborlist<RealType>::Y() const {
  // Doesn't scale with the number of systems
  return ceil_divide(this->num_column_blocks(), WARP_SIZE);
};

template <typename RealType>
int Neighborlist<RealType>::num_row_blocks() const {
  return ceil_divide(NR_, TILE_SIZE);
}

// max_ixn_count determines the number of tile-atom interaction counts. For each
// tile that interacts with another it can have TILE_SIZE tile-atom
// interactions. Note that d_ixn_count_ is only the number of tile-tile
// interactions, and differs by a factor of TILE_SIZE
template <typename RealType> int Neighborlist<RealType>::max_ixn_count() const {
  // The maximum number of tile-atom interactions, equal to # tile-tile
  // interactions multiplied by TILE_SIZE (typically 32). Use the maximum value
  // of N to compute the size of the upper triangular matrix to support any set
  // of row indices.
  const int n_blocks = ceil_divide(max_size_, TILE_SIZE);
  int max_tile_tile_interactions = (n_blocks * (n_blocks + 1)) / 2;
  // Each tile-tile interaction can have TILE_SIZE tile-atom interactions
  return max_tile_tile_interactions * TILE_SIZE;
}

template class Neighborlist<double>;
template class Neighborlist<float>;

} // namespace tmd
