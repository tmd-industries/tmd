// Copyright 2019-2025, Relay Therapeutics
// Modifications Copyright 2025 Forrest York
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "assert.h"
#include "gpu_utils.cuh"
#include "k_periodic_torsion.cuh"
#include "kernel_utils.cuh"
#include "math_utils.cuh"
#include "periodic_torsion.hpp"
#include <vector>

namespace tmd {

template <typename RealType>
PeriodicTorsion<RealType>::PeriodicTorsion(
    const int num_batches, const int num_atoms,
    const std::vector<int> &torsion_idxs, // [A, 4]
    const std::vector<int> &system_idxs   // [A]
    )
    : num_batches_(num_batches), num_atoms_(num_atoms),
      max_idxs_(torsion_idxs.size() / IDXS_DIM), cur_num_idxs_(max_idxs_),
      nrg_accum_(num_batches_, cur_num_idxs_),
      kernel_ptrs_({// enumerate over every possible kernel combination
                    // U: Compute U
                    // X: Compute DU_DX
                    // P: Compute DU_DP              U  X  P
                    &k_periodic_torsion<RealType, 3, 0, 0, 0>,
                    &k_periodic_torsion<RealType, 3, 0, 0, 1>,
                    &k_periodic_torsion<RealType, 3, 0, 1, 0>,
                    &k_periodic_torsion<RealType, 3, 0, 1, 1>,
                    &k_periodic_torsion<RealType, 3, 1, 0, 0>,
                    &k_periodic_torsion<RealType, 3, 1, 0, 1>,
                    &k_periodic_torsion<RealType, 3, 1, 1, 0>,
                    &k_periodic_torsion<RealType, 3, 1, 1, 1>}) {

  if (torsion_idxs.size() % IDXS_DIM != 0) {
    throw std::runtime_error("torsion_idxs.size() must be exactly " +
                             std::to_string(IDXS_DIM) + "*k");
  }
  if (system_idxs.size() != max_idxs_) {
    throw std::runtime_error("system_idxs.size() != (torsion_idxs.size() / " +
                             std::to_string(IDXS_DIM) + "), got " +
                             std::to_string(system_idxs.size()) + " and " +
                             std::to_string(max_idxs_));
  }

  for (int a = 0; a < cur_num_idxs_; a++) {
    auto i = torsion_idxs[a * IDXS_DIM + 0];
    auto j = torsion_idxs[a * IDXS_DIM + 1];
    auto k = torsion_idxs[a * IDXS_DIM + 2];
    auto l = torsion_idxs[a * IDXS_DIM + 3];
    if (i == j || i == k || i == l || j == k || j == l || k == l) {
      throw std::runtime_error("torsion quads must be unique");
    }
  }

  cudaSafeMalloc(&d_torsion_idxs_,
                 cur_num_idxs_ * IDXS_DIM * sizeof(*d_torsion_idxs_));
  gpuErrchk(hipMemcpy(d_torsion_idxs_, &torsion_idxs[0],
                       cur_num_idxs_ * IDXS_DIM * sizeof(*d_torsion_idxs_),
                       hipMemcpyHostToDevice));

  cudaSafeMalloc(&d_u_buffer_, cur_num_idxs_ * sizeof(*d_u_buffer_));

  cudaSafeMalloc(&d_system_idxs_, cur_num_idxs_ * sizeof(*d_system_idxs_));
  gpuErrchk(hipMemcpy(d_system_idxs_, &system_idxs[0],
                       cur_num_idxs_ * sizeof(*d_system_idxs_),
                       hipMemcpyHostToDevice));
};

template <typename RealType> PeriodicTorsion<RealType>::~PeriodicTorsion() {
  gpuErrchk(hipFree(d_torsion_idxs_));
  gpuErrchk(hipFree(d_u_buffer_));
  gpuErrchk(hipFree(d_system_idxs_));
};

template <typename RealType>
void PeriodicTorsion<RealType>::execute_device(
    const int batches, const int N, const int P, const RealType *d_x,
    const RealType *d_p, const RealType *d_box, unsigned long long *d_du_dx,
    unsigned long long *d_du_dp, __int128 *d_u, hipStream_t stream) {

  const int tpb = DEFAULT_THREADS_PER_BLOCK;
  const int blocks = ceil_divide(cur_num_idxs_, tpb);

  if (blocks > 0) {
    if (P != 3 * cur_num_idxs_) {
      throw std::runtime_error("PeriodicTorsion::execute_device(): expected P "
                               "== 3*cur_num_idxs_, got P=" +
                               std::to_string(P) + ", 3*cur_num_idxs_=" +
                               std::to_string(3 * cur_num_idxs_));
    }

    int kernel_idx = 0;
    kernel_idx |= d_du_dp ? 1 << 0 : 0;
    kernel_idx |= d_du_dx ? 1 << 1 : 0;
    kernel_idx |= d_u ? 1 << 2 : 0;

    kernel_ptrs_[kernel_idx]<<<blocks, tpb, 0, stream>>>(
        num_atoms_, cur_num_idxs_, d_x, d_box, d_p, d_torsion_idxs_,
        d_system_idxs_, d_du_dx, d_du_dp,
        d_u == nullptr ? nullptr : d_u_buffer_);
    gpuErrchk(hipPeekAtLastError());

    if (d_u) {
      // nullptr for the d_system_idxs as batch size is fixed to 1
      nrg_accum_.sum_device(cur_num_idxs_, d_u_buffer_, d_system_idxs_, d_u,
                            stream);
    }
  }
};

template <typename RealType>
void PeriodicTorsion<RealType>::set_idxs_device(const int num_idxs,
                                                const int *d_new_idxs,
                                                hipStream_t stream) {
  if (max_idxs_ < num_idxs) {
    throw std::runtime_error("set_idxs_device(): Max number of torsions " +
                             std::to_string(max_idxs_) +
                             " is less than new idxs " +
                             std::to_string(num_idxs));
  }
  gpuErrchk(hipMemcpyAsync(d_torsion_idxs_, d_new_idxs,
                            num_idxs * IDXS_DIM * sizeof(*d_torsion_idxs_),
                            hipMemcpyDeviceToDevice, stream));
  cur_num_idxs_ = num_idxs;
}

template <typename RealType>
int PeriodicTorsion<RealType>::get_num_idxs() const {
  return cur_num_idxs_;
}

template <typename RealType> int *PeriodicTorsion<RealType>::get_idxs_device() {
  return d_torsion_idxs_;
}

template <typename RealType>
std::vector<int> PeriodicTorsion<RealType>::get_idxs_host() const {
  return device_array_to_vector<int>(cur_num_idxs_ * IDXS_DIM, d_torsion_idxs_);
}

template <typename RealType> int PeriodicTorsion<RealType>::batch_size() const {
  return num_batches_;
}

template class PeriodicTorsion<double>;
template class PeriodicTorsion<float>;

} // namespace tmd
