// Copyright 2019-2025, Relay Therapeutics
// Modifications Copyright 2025 Forrest York
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "assert.h"
#include "fixed_point.hpp"
#include "gpu_utils.cuh"
#include "nonbonded_common.hpp"
#include "summed_potential.hpp"
#include <memory>
#include <numeric>
#include <stdexcept>

namespace tmd {

template <typename RealType>
SummedPotential<RealType>::SummedPotential(
    const std::vector<std::shared_ptr<Potential<RealType>>> potentials,
    const std::vector<int> params_sizes, const bool parallel)
    : potentials_(potentials), params_sizes_(params_sizes),
      P_(std::accumulate(params_sizes.begin(), params_sizes.end(), 0)),
      parallel_(parallel), d_u_buffer_(potentials.size()),
      nrg_accum_(1, potentials_.size()) {
  if (potentials_.size() != params_sizes_.size()) {
    throw std::runtime_error(
        "number of potentials != number of parameter sizes");
  }
};

template <typename RealType> SummedPotential<RealType>::~SummedPotential(){};

template <typename RealType>
const std::vector<std::shared_ptr<Potential<RealType>>> &
SummedPotential<RealType>::get_potentials() {
  return potentials_;
}

template <typename RealType>
const std::vector<int> &SummedPotential<RealType>::get_parameter_sizes() {
  return params_sizes_;
}

template <typename RealType>
void SummedPotential<RealType>::execute_device(
    const int batches, const int N, const int P, const RealType *d_x,
    const RealType *d_p, const RealType *d_box, unsigned long long *d_du_dx,
    unsigned long long *d_du_dp, __int128 *d_u, hipStream_t stream) {

  assert(batches == 1);
  if (P != P_) {
    throw std::runtime_error(
        "SummedPotential<RealType>::execute_device(): expected " +
        std::to_string(P_) + " parameters, got " + std::to_string(P));
  }
  if (d_u) {
    gpuErrchk(hipMemsetAsync(d_u_buffer_.data, 0, d_u_buffer_.size(), stream));
  }

  int offset = 0;
  if (parallel_) {
    manager_.record_master_event(stream);
    for (auto i = 0; i < potentials_.size(); i++) {
      // Always sync the new streams with the incoming stream to ensure that the
      // state of the incoming buffers are valid
      manager_.wait_on_master(i, stream);
    }
  }
  hipStream_t pot_stream = stream;
  for (auto i = 0; i < potentials_.size(); i++) {
    if (parallel_) {
      pot_stream = manager_.get_stream(i);
    }
    potentials_[i]->execute_device(
        batches, N, params_sizes_[i], d_x, d_p + offset, d_box, d_du_dx,
        d_du_dp == nullptr ? nullptr : d_du_dp + offset,
        d_u == nullptr ? nullptr : d_u_buffer_.data + i, pot_stream);
    offset += params_sizes_[i];
  }
  if (parallel_) {
    for (auto i = 0; i < potentials_.size(); i++) {
      manager_.record_and_wait_on_child(i, stream);
    }
  }
  if (d_u) {
    // nullptr for the d_system_idxs as batch size is fixed to 1
    nrg_accum_.sum_device(potentials_.size(), d_u_buffer_.data, nullptr, d_u,
                          stream);
  }
};

template <typename RealType>
void SummedPotential<RealType>::du_dp_fixed_to_float(
    const int N, const int P, const unsigned long long *du_dp,
    RealType *du_dp_float) {

  int offset = 0;

  for (auto i = 0; i < potentials_.size(); i++) {
    potentials_[i]->du_dp_fixed_to_float(N, params_sizes_[i], du_dp + offset,
                                         du_dp_float + offset);
    offset += params_sizes_[i];
  }
}

template class SummedPotential<double>;
template class SummedPotential<float>;

} // namespace tmd
